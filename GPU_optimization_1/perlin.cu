#include "hip/hip_runtime.h"
#include <hip/hip_math_constants.h>

// Timing function
double CLOCK() {
    struct timespec t;
    clock_gettime(CLOCK_MONOTONIC,  &t);
    return (t.tv_sec * 1000)+(t.tv_nsec*1e-6);
}



// CUDA kernel to precompute the grid gratients
__global__ void generateGridGradients(float4* gradients, int perlinGridSize, unsigned int seed) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x < perlinGridSize && y < perlinGridSize) {
        // Create a pseudo-random hash from the index and seed
        unsigned int idx = y * perlinGridSize + x;
        unsigned int hash = seed;
        hash ^= x * 1523 + y * 823;
        hash *= 0x85ebca6b;
        hash ^= hash >> 13;
        hash *= 0xc2b2ae35;
        hash ^= hash >> 16;
        
        // Generate a unit vector in a random direction
        float angle = hash * (2.0f * HIP_PI_F / UINT_MAX);
        
        // Store as normalized vector components
        gradients[idx].x = __sinf(angle);   // CUDA sin/cos intrinsics
        gradients[idx].y = __cosf(angle);
        gradients[idx].z = 0.0f;
    }
}

// Interpolate between two values using smoothstep interpolation
__device__ float interpolate(float a0, float a1, float w) {
    return (a1 - a0) * (3.0f - w * 2.0f) * w * w + a0;
}

// Compute the Perlin noise value at a given point (x, y) using the specified seed
__device__ float pixelPerlin(float x, float y, hipTextureObject_t gradientTexObj) {
    int x0 = (int)x;
    int y0 = (int)y;
    int x1 = x0 + 1;
    int y1 = y0 + 1;

    float sx = x - (float)x0;
    float sy = y - (float)y0;

    // Convert to normalized coordinates for the texture
    float u0 = (float)x0;// / perlinGridSize;
    float v0 = (float)y0;// / perlinGridSize;
    float u1 = (float)x1;// / perlinGridSize;
    float v1 = (float)y1;// / perlinGridSize;

    // Read the gradient vectors from the texture memory using the texture object
    float4 g00 = tex2D<float4>(gradientTexObj, u0, v0);
    float4 g10 = tex2D<float4>(gradientTexObj, u1, v0);
    float4 g01 = tex2D<float4>(gradientTexObj, u0, v1);
    float4 g11 = tex2D<float4>(gradientTexObj, u1, v1);

    // Compute the dot products between the gradients and the distance vectors
    float dx0 = x - (float)x0;
    float dy0 = y - (float)y0;
    float dx1 = x - (float)x1;
    float dy1 = y - (float)y1;

    float n00 = g00.x * dx0 + g00.y * dy0;
    float n10 = g10.x * dx1 + g10.y * dy0;
    float n01 = g01.x * dx0 + g01.y * dy1;
    float n11 = g11.x * dx1 + g11.y * dy1;

    // Interpolate along x
    float ix0 = interpolate(n00, n10, sx);
    float ix1 = interpolate(n01, n11, sx);

    // Interpolate along y
    return interpolate(ix0, ix1, sy);
}

// CUDA kernel to generate Perlin noise 
__global__ void perlinKernel(float* d_image, int windowWidth, int windowHeight, int perlinGridSize, int numOctaves, unsigned seed, hipTextureObject_t gradientTexObj) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if the pixel is within bounds
    if (x >= windowWidth || y >= windowHeight) return;

    // Initialize the noise, amplitude, and frequency
    float noise = 0.0f;
    float freq = 1.0f;
    float amp = 1.0f;

    // Loop through the octaves to accumulate the noise value
    for (int i = 0; i < numOctaves; i++) {
        noise += amp * pixelPerlin(x * freq / perlinGridSize, y * freq / perlinGridSize, gradientTexObj);
        freq *= 2.0f; // Increase frequency for the next octave
        amp /= 2.0f; // Decrease amplitude for the next octave
    }

    // Normalize the noise value to the range [0, 1]
    noise = fminf(1.0f, fmaxf(-1.0f, noise)); // Clamp the value to [-1, 1]

    d_image[y * windowWidth + x] = noise; // Write the noise to the global array
}

// Build the Perlin Noise heightmap
double buildPerlinNoise(int windowWidth, int windowHeight, int perlinGridSize, int numOctaves, unsigned seed, float** outImage) {
    // Timing
    double start, finish;
    
    /** Generate gradients for perlin noise */
    // Allocate device memory
    float4 *d_gradients;
    hipMalloc(&d_gradients, perlinGridSize * perlinGridSize * sizeof(float4));
    
    // Device kernel configuration
    dim3 blockSize(16, 16);
    dim3 gradientGridDim((perlinGridSize + blockSize.x - 1) / blockSize.x,
                 (perlinGridSize + blockSize.y - 1) / blockSize.y);
    
    // Call device kernel
    start = CLOCK();
    generateGridGradients<<<gradientGridDim, blockSize>>>(d_gradients, perlinGridSize, seed);
    
    // Wait for kernel to finish
    hipDeviceSynchronize();
    
    // Move result to texture memory:
    
    // 1. Create a CUDA array for the texture data
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();
    hipArray* gradientArray;
    hipMallocArray(&gradientArray, &channelDesc, perlinGridSize, perlinGridSize);
    
    // 2. Copy the gradient data from device memory to the CUDA array
    hipMemcpy2DToArray(gradientArray, 0, 0, 
                        d_gradients, perlinGridSize * sizeof(float4),
                        perlinGridSize * sizeof(float4), perlinGridSize,
                        hipMemcpyDeviceToDevice);
    
    // 3. Set up the texture resource descriptor
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = gradientArray;
    
    // 4. Set up the texture descriptor
    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeWrap;  // Wrap addressing for Perlin noise
    texDesc.addressMode[1] = hipAddressModeWrap;
    texDesc.filterMode = hipFilterModeLinear;     // Bilinear interpolation
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 1;                  // Use normalized coordinates [0,1]
    
    // 5. Create the texture object
    hipTextureObject_t gradientTexObj = 0;
    hipCreateTextureObject(&gradientTexObj, &resDesc, &texDesc, NULL);
    
    // Remember to clean up the texture object and array when done
    // hipDestroyTextureObject(gradientTexObj);
    // hipFreeArray(gradientArray);
    // hipFree(d_gradients);

    // Amt of data to copy back and forth
    int bytes = windowWidth * windowHeight * sizeof(float);

    // Initialize and allocate host + device memory
    float* h_image = (float*)malloc(bytes);
    float* d_image;
    hipMalloc(&d_image, bytes);

    // device kernel launch config
    dim3 noiseGridDim((windowWidth + 15) / 16, (windowHeight + 15) / 16);

    // Launch device kernel
    start = CLOCK();
    perlinKernel<<<noiseGridDim, blockSize>>>(d_image, windowWidth, windowHeight, 
        perlinGridSize, numOctaves, seed, 
        gradientTexObj);

    // Bring the data back to the host
    hipMemcpy(h_image, d_image, windowWidth * windowHeight * sizeof(float), hipMemcpyDeviceToHost);
    finish = CLOCK();
    
    // Convert 1D to 2D
    *outImage = new float[windowWidth * windowHeight];
    memcpy(*outImage, h_image, windowWidth * windowHeight * sizeof(float));

    // Cleanup
    hipDestroyTextureObject(gradientTexObj);
    hipFreeArray(gradientArray);
    hipFree(d_gradients);
    hipFree(d_image);
    free(h_image);

    return (finish - start);
}

